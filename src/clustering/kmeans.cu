#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <helper/helper.cuh>
#include <helper/helperCUDA.h>

__global__
void determineMembershipKernel(const float* X, const float* centers, int* membership, const int numSamples, const int numFeatures, const int numCenters) {
    // Each thread is responsible for each data point
    int idxSample = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory to accelerate computation
    // We copy X and centers to shared memory to avoid unnecessary global memory access
    extern __shared__ float sharedMem[];

    float* sharedX = sharedMem;
    // Each block has blockDim.x threads, and each threads is responsible for one data point
    // Therefore, in each block, the shared memory allocated for X is blockDim.x * numFeatures in total
    const int numSamplesThisBlock = blockDim.x;
    float* sharedCenters = sharedMem + (numSamplesThisBlock * numFeatures);
    
    const int idxSampleSharedMem = threadIdx.x;

    // The first 'numCenters' threads in this block are responsible for loading the data of centers
    if (threadIdx.x < numCenters) {
        for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
            sharedCenters[idxFeature * numCenters + threadIdx.x] = centers[idxFeature * numCenters + threadIdx.x];
        }
    }
    __syncthreads();

    while (idxSample < numSamples) {
        // Load this block's data into the shared memory
        for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
            sharedX[idxFeature * numSamplesThisBlock + idxSampleSharedMem] = X[idxFeature * numSamples + idxSample];
        }
        __syncthreads();

        float minDist = FLT_MAX;
        int minDistIdx = -1;
        for (int idxCenter = 0; idxCenter < numCenters; idxCenter++) {
            float dist = 0;
            for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
                dist += pow(sharedX[idxFeature * numSamplesThisBlock + idxSampleSharedMem] - sharedCenters[idxFeature * numCenters + idxCenter], 2);
            }
            if (minDistIdx == -1 || minDist > dist) {
                minDist = dist;
                minDistIdx = idxCenter;
            }
        }
        membership[idxSample] = minDistIdx;
        idxSample += gridDim.x * blockDim.x;
    }
}

__global__
void updateCentersKernel(const float* X, const int* membership, float* centers, int* deviceSamplesCount, const int idxCenter, const int numSamples, const int numFeatures, const int numCenters) {
    // For updating every center, this kernel will be invoked once
    // In this kernel, we perform a reduction sum
    int idxSample = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ float sharedMem[];

    float* sharedX = sharedMem;
    const int numSamplesSharedMem = blockDim.x;
    int* sharedSampleCount = (int*)(sharedMem + (numSamplesSharedMem * numFeatures));

    const int idxSampleSharedMem = threadIdx.x;

    for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
        sharedX[idxFeature * numSamplesSharedMem + idxSampleSharedMem] = 0;
    }
    sharedSampleCount[idxSampleSharedMem] = 0;
    __syncthreads();

    while (idxSample < numSamples) {
        // Initialize the shared memory
        int member = membership[idxSample];
        sharedSampleCount[idxSampleSharedMem] = member == idxCenter;
        for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
            sharedX[idxFeature * numSamplesSharedMem + idxSampleSharedMem] = X[idxFeature * numSamples + idxSample] * (member == idxCenter);
        }
        __syncthreads();

        // Reduction begin here
        int range = blockDim.x;
        for (int i = 0; i < (int)log2((float)blockDim.x); i++) {
            range /= 2;
            if (idxSampleSharedMem < range) {
                sharedSampleCount[idxSampleSharedMem] += sharedSampleCount[idxSampleSharedMem + range];
                sharedSampleCount[idxSampleSharedMem + range] = 0;
                for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
                    sharedX[idxFeature * numSamplesSharedMem + idxSampleSharedMem] += sharedX[idxFeature * numSamplesSharedMem + (idxSampleSharedMem + range)];
                    sharedX[idxFeature * numSamplesSharedMem + (idxSampleSharedMem + range)] = 0;
                }
            }
            __syncthreads();
        }

        // Use atomic operation to accumulate the result in global memory
        if (threadIdx.x == 0) {
            atomicAdd(deviceSamplesCount, sharedSampleCount[0]);
            sharedSampleCount[0] = 0;
            for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
                atomicAdd(centers + (idxFeature * numCenters + idxCenter), sharedX[idxFeature * numSamplesSharedMem]);
                sharedX[idxFeature * numSamplesSharedMem] = 0;
            }
        }

        idxSample += gridDim.x * blockDim.x;
    }
}

// __global__
// void updateCentersKernel(const float* X, const int* membership, float* centers, int* numSamplesThisCenter, const int numSamples, const int numFeatures, const int numCenters) {
//     /*
//      * Pre-condition: centers and numSamplesThisCenter are initialized to all zeros
//      */
//     int idxCenter = threadIdx.x + blockIdx.x * blockDim.x;
    
//     while (idxCenter < numCenters) {

//         for (int i = 0; i < numSamples; i++) {
//             if (membership[i] == idxCenter) {
//                 for (int j = 0; j < numFeatures; j++) {
//                     centers[j * numCenters + idxCenter] += X[j * numSamples + i];
//                 }
//                 numSamplesThisCenter[idxCenter]++;
//             }
//         }

//         for (int j = 0; j < numFeatures; j++) {
//             centers[j * numCenters + idxCenter] /= numSamplesThisCenter[idxCenter];
//         }

//         idxCenter += gridDim.x * blockDim.x;
//     }
// }

/* GPU version of KMeans */
void _kmeansGPU(const float* X, const float* initCenters, const int numSamples, const int numFeatures, const int numCenters, const int maxNumIteration, const float tolerance, float* centers, int* membership, int* numIterations) {
    /*
     * Use GPU to accelerate the KMeans algorithm
     * The whole process will be done using three separate kernels:
     *  1. The first kernel compute the distance of points to clusters
     *  2  The second kernel determine the membership of points to clusters
     *  3. The third kernel update the center of each cluster
     */

    assert(maxNumIteration >= 0);

    memcpy(centers, initCenters, sizeof(float) * numCenters * numFeatures);
    bool endFlag = maxNumIteration == 0;
    int iterationCount = 0;

    // Malloc space on GPU
    float* deviceX;
    float* deviceCenters;
    int* deviceMembership;
    float* deviceOldCenters;
    int* deviceSamplesCount;
    // These are temporary array here for transpose
    float* tempDeviceX;
    float* tempDeviceCenters;

    CUDA_CALL( hipMalloc(&deviceX, sizeof(float) * numSamples * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceCenters, sizeof(float) * numCenters * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceMembership, sizeof(int) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceOldCenters, sizeof(float) * numCenters * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceSamplesCount, sizeof(int)) );
    CUDA_CALL( hipMalloc(&tempDeviceX, sizeof(float) * numSamples * numFeatures) );
    CUDA_CALL( hipMalloc(&tempDeviceCenters, sizeof(float) * numCenters * numFeatures) );

    // Malloc space on CPU
    int* samplesCount = (int*)malloc(sizeof(int));

    // Determine the block width
    const int BLOCKWIDTH = 1024;

    // Initialize the cublas handle
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    // These are useful when calling cublas functions
    float one = 1.0, zero = 0.0, negOne = -1.0;

    // Transpose deviceX, deviceCenters here to enable coalesced memory access in the kernel
    CUDA_CALL( hipMemcpy(tempDeviceX, X, sizeof(float) * numSamples * numFeatures, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(tempDeviceCenters, centers, sizeof(float) * numCenters * numFeatures, hipMemcpyHostToDevice) );

    CUBLAS_CALL( hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, numSamples, numFeatures, &one, tempDeviceX, numFeatures, &zero, tempDeviceX, numSamples, deviceX, numSamples) );
    CUBLAS_CALL( hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, numCenters, numFeatures, &one, tempDeviceCenters, numFeatures, &zero, tempDeviceCenters, numCenters, deviceCenters, numCenters) );

    while (!endFlag) {        
        // Determine the membership of each sample
        int numBlock = min(65535, ((numSamples) + BLOCKWIDTH - 1) / BLOCKWIDTH);
        int numBytesSharedMemory = BLOCKWIDTH * sizeof(float) * numFeatures + sizeof(float) * numCenters * numFeatures;        
        determineMembershipKernel<<<numBlock, BLOCKWIDTH, numBytesSharedMemory>>>(deviceX, deviceCenters, deviceMembership, numSamples, numFeatures, numCenters);;  

        // Save the result of old centers
        CUDA_CALL( hipMemcpy(deviceOldCenters, deviceCenters, sizeof(float) * numCenters * numFeatures, hipMemcpyDeviceToDevice) );
        CUDA_CALL( hipMemset(deviceCenters, 0, sizeof(float) * numCenters * numFeatures));

        // Update the center estimation
        numBlock = min(65535, ((numSamples) + BLOCKWIDTH - 1) / BLOCKWIDTH);
        numBytesSharedMemory = BLOCKWIDTH * sizeof(float) * numFeatures + BLOCKWIDTH * sizeof(float);

        for (int idxCenter = 0; idxCenter < numCenters; idxCenter++) {
            CUDA_CALL( hipMemset(deviceSamplesCount, 0, sizeof(int)) );
            updateCentersKernel<<<numBlock, BLOCKWIDTH, numBytesSharedMemory>>>(deviceX, deviceMembership, deviceCenters, deviceSamplesCount, idxCenter, numSamples, numFeatures, numCenters);
            CUDA_CALL( hipMemcpy(samplesCount, deviceSamplesCount, sizeof(int), hipMemcpyDeviceToHost) );
            float scale = 1.0 / (*samplesCount);
            CUBLAS_CALL( hipblasSscal(cublasHandle, numFeatures, &scale, deviceCenters + idxCenter, numCenters) );
        }

        // Test for coverage
        iterationCount++;
        if (iterationCount >= maxNumIteration) {
            endFlag = true;
        } else {
            // Careful here, cuBlas assumes column major storage
            // Perform element-wise subtraction
            CUBLAS_CALL( hipblasSgeam(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, numCenters, numFeatures, &one, deviceOldCenters, numCenters, &negOne, deviceCenters, numCenters, deviceOldCenters, numCenters) );
            float diff = 0.0;
            // Compute the F-norm
            CUBLAS_CALL( hipblasSdot(cublasHandle, numCenters * numFeatures, deviceOldCenters, 1, deviceOldCenters, 1, &diff) );
            endFlag = sqrt(diff) < tolerance;
        }
    }

    // Copy the result back to host
    // Tranpose the deviceCenters back
    CUBLAS_CALL( hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, numFeatures, numCenters, &one, deviceCenters, numCenters, &zero, deviceCenters, numFeatures, tempDeviceCenters, numFeatures) );
    float* temp = tempDeviceCenters;
    tempDeviceCenters = deviceCenters;
    deviceCenters = temp;

    CUDA_CALL( hipMemcpy(centers, deviceCenters, sizeof(float) * numCenters * numFeatures, hipMemcpyDeviceToHost) );
    CUDA_CALL( hipMemcpy(membership, deviceMembership, sizeof(int) * numSamples, hipMemcpyDeviceToHost) );
    *numIterations = iterationCount;

    // Free all resources on GPU
    CUDA_CALL( hipFree(deviceX) );
    CUDA_CALL( hipFree(deviceCenters) );
    CUDA_CALL( hipFree(deviceMembership) );
    CUDA_CALL( hipFree(deviceOldCenters) );
    CUDA_CALL( hipFree(deviceSamplesCount) );
    CUDA_CALL( hipFree(tempDeviceCenters) );
    CUDA_CALL( hipFree(tempDeviceX) );

    // Free resources on CPU
    if (samplesCount) {
        free(samplesCount);
    }

    CUDA_CALL( hipblasDestroy(cublasHandle) );
    return;
}