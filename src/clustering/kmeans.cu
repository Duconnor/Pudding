#include "hip/hip_runtime.h"
#include <assert.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <helper/helper.cuh>
#include <helper/helper_CUDA.h>

__global__
void determineMembershipKernel(const float* X, const float* centers, int* membership, const int numSamples, const int numFeatures, const int numCenters) {
    // Each thread is responsible for each data point
    int idxSample = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory to accelerate computation
    // We copy X and centers to shared memory to avoid unnecessary global memory access
    extern __shared__ float sharedMem[];

    float* sharedX = sharedMem;
    // Each block has blockDim.x threads, and each threads is responsible for one data point
    // Therefore, in each block, the shared memory allocated for X is blockDim.x * numFeatures in total
    const int numSamplesThisBlock = blockDim.x;
    float* sharedCenters = sharedMem + (numSamplesThisBlock * numFeatures);
    
    const int idxSampleSharedMem = threadIdx.x;

    // The first 'numCenters' threads in this block are responsible for loading the data of centers
    // TODO: We can't just load all centers into the shared memory, there wouldn't be enough space in some extreme case.
    int sharedCentersIdx = threadIdx.x;
    while (sharedCentersIdx < numCenters) {
        for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
            sharedCenters[idxFeature * numCenters + sharedCentersIdx] = centers[idxFeature * numCenters + sharedCentersIdx];
        }
        sharedCentersIdx += blockDim.x;
    }
    __syncthreads();

    while (idxSample < numSamples) {
        // Load this block's data into the shared memory
        for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
            sharedX[idxFeature * numSamplesThisBlock + idxSampleSharedMem] = X[idxFeature * numSamples + idxSample];
        }
        __syncthreads();

        float minDist = FLT_MAX;
        int minDistIdx = -1;
        for (int idxCenter = 0; idxCenter < numCenters; idxCenter++) {
            float dist = 0;
            for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
                dist += pow(sharedX[idxFeature * numSamplesThisBlock + idxSampleSharedMem] - sharedCenters[idxFeature * numCenters + idxCenter], 2);
            }
            if (minDistIdx == -1 || minDist > dist) {
                minDist = dist;
                minDistIdx = idxCenter;
            }
        }
        membership[idxSample] = minDistIdx;
        idxSample += gridDim.x * blockDim.x;
    }
}

__global__
void updateCentersKernel(const float* X, const int* membership, float* centers, int* deviceSamplesCount, const int idxCenter, const int numSamples, const int numFeatures, const int numCenters) {
    // For updating every center, this kernel will be invoked once
    // In this kernel, we perform a reduction sum
    int idxSample = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ float sharedMem[];

    float* sharedX = sharedMem;
    const int numSamplesSharedMem = blockDim.x;
    int* sharedSampleCount = (int*)(sharedMem + (numSamplesSharedMem * numFeatures));

    const int idxSampleSharedMem = threadIdx.x;

    for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
        sharedX[idxFeature * numSamplesSharedMem + idxSampleSharedMem] = 0;
    }
    sharedSampleCount[idxSampleSharedMem] = 0;
    __syncthreads();

    while (idxSample < numSamples) {
        // Initialize the shared memory
        int member = membership[idxSample];
        sharedSampleCount[idxSampleSharedMem] = member == idxCenter;
        for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
            sharedX[idxFeature * numSamplesSharedMem + idxSampleSharedMem] = X[idxFeature * numSamples + idxSample] * (member == idxCenter);
        }
        __syncthreads();

        // Reduction begin here
        int range = blockDim.x;
        for (int i = 0; i < (int)log2((float)blockDim.x); i++) {
            range /= 2;
            if (idxSampleSharedMem < range) {
                sharedSampleCount[idxSampleSharedMem] += sharedSampleCount[idxSampleSharedMem + range];
                sharedSampleCount[idxSampleSharedMem + range] = 0;
                for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
                    sharedX[idxFeature * numSamplesSharedMem + idxSampleSharedMem] += sharedX[idxFeature * numSamplesSharedMem + (idxSampleSharedMem + range)];
                    sharedX[idxFeature * numSamplesSharedMem + (idxSampleSharedMem + range)] = 0;
                }
            }
            __syncthreads();
        }

        // Use atomic operation to accumulate the result in global memory
        if (threadIdx.x == 0) {
            atomicAdd(deviceSamplesCount, sharedSampleCount[0]);
            sharedSampleCount[0] = 0;
            for (int idxFeature = 0; idxFeature < numFeatures; idxFeature++) {
                atomicAdd(centers + (idxFeature * numCenters + idxCenter), sharedX[idxFeature * numSamplesSharedMem]);
                sharedX[idxFeature * numSamplesSharedMem] = 0;
            }
        }

        idxSample += gridDim.x * blockDim.x;
    }
}

/* GPU version of KMeans */
void _kmeansGPU(const float* X, const float* initCenters, const int numSamples, const int numFeatures, const int numCenters, const int maxNumIteration, const float tolerance, float* centers, int* membership, int* numIterations) {
    /*
     * Use GPU to accelerate the KMeans algorithm
     * The whole process will be done using two separate kernels:
     *  1. The first kernel determine the membership of points to clusters
     *  2. The second kernel update the center of each cluster
     */

    // TODO: Maybe switch to the transposeMatrix helper function for performing transpose.

    assert(maxNumIteration >= 0);

    memcpy(centers, initCenters, sizeof(float) * numCenters * numFeatures);
    bool endFlag = maxNumIteration == 0;
    int iterationCount = 0;

    // Malloc space on GPU
    float* deviceX;
    float* deviceCenters;
    int* deviceMembership;
    float* deviceOldCenters;
    int* deviceSamplesCount;

    CUDA_CALL( hipMalloc(&deviceX, sizeof(float) * numSamples * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceCenters, sizeof(float) * numCenters * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceMembership, sizeof(int) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceOldCenters, sizeof(float) * numCenters * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceSamplesCount, sizeof(int)) );
    CUDA_CALL( hipMemcpy(deviceX, X, sizeof(float) * numSamples * numFeatures, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceCenters, centers, sizeof(float) * numCenters * numFeatures, hipMemcpyHostToDevice) );

    // Malloc space on CPU
    int* samplesCount = (int*)malloc(sizeof(int));

    // Determine the block width
    const int BLOCKWIDTH = 1024;

    // Initialize the cublas handle
    hipblasHandle_t cublasHandle;
    CUBLAS_CALL( hipblasCreate(&cublasHandle) );
    // These are useful when calling cublas functions
    float one = 1.0, negOne = -1.0;

    // Transpose deviceX, deviceCenters here to enable coalesced memory access in the kernel
    transposeMatrix(deviceX, numSamples, numFeatures);
    transposeMatrix(deviceCenters, numCenters, numFeatures);

    while (!endFlag) {        
        // Determine the membership of each sample
        int numBlock = min(65535, ((numSamples) + BLOCKWIDTH - 1) / BLOCKWIDTH);
        int numBytesSharedMemory = BLOCKWIDTH * sizeof(float) * numFeatures + sizeof(float) * numCenters * numFeatures;
        if (numBytesSharedMemory > MAXSHAREDMEMBYTES) {
            assert(false && "No enough shared memory");
        }
        determineMembershipKernel<<<numBlock, BLOCKWIDTH, numBytesSharedMemory>>>(deviceX, deviceCenters, deviceMembership, numSamples, numFeatures, numCenters);;  

        // Save the result of old centers
        CUDA_CALL( hipMemcpy(deviceOldCenters, deviceCenters, sizeof(float) * numCenters * numFeatures, hipMemcpyDeviceToDevice) );
        CUDA_CALL( hipMemset(deviceCenters, 0, sizeof(float) * numCenters * numFeatures));

        // Update the center estimation
        numBlock = min(65535, ((numSamples) + BLOCKWIDTH - 1) / BLOCKWIDTH);
        numBytesSharedMemory = BLOCKWIDTH * sizeof(float) * numFeatures + BLOCKWIDTH * sizeof(float);
        if (numBytesSharedMemory > MAXSHAREDMEMBYTES) {
            assert(false && "No enough shared memory");
        }

        for (int idxCenter = 0; idxCenter < numCenters; idxCenter++) {
            CUDA_CALL( hipMemset(deviceSamplesCount, 0, sizeof(int)) );
            updateCentersKernel<<<numBlock, BLOCKWIDTH, numBytesSharedMemory>>>(deviceX, deviceMembership, deviceCenters, deviceSamplesCount, idxCenter, numSamples, numFeatures, numCenters);
            CUDA_CALL( hipMemcpy(samplesCount, deviceSamplesCount, sizeof(int), hipMemcpyDeviceToHost) );
            if (*samplesCount == 0) {
                // Empty cluster, we keep it unchanged
                CUBLAS_CALL( hipblasScopy(cublasHandle, numFeatures, deviceOldCenters + idxCenter, numCenters, deviceCenters + idxCenter, numCenters) );
            } else {
                float scale = 1.0 / (*samplesCount);
                CUBLAS_CALL( hipblasSscal(cublasHandle, numFeatures, &scale, deviceCenters + idxCenter, numCenters) );
            }
        }

        // Test for coverage
        iterationCount++;
        if (iterationCount >= maxNumIteration) {
            endFlag = true;
        } else {
            // Careful here, cuBlas assumes column major storage
            // Perform element-wise subtraction
            CUBLAS_CALL( hipblasSgeam(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, numCenters, numFeatures, &one, deviceOldCenters, numCenters, &negOne, deviceCenters, numCenters, deviceOldCenters, numCenters) );
            float diff = 0.0;
            // Compute the F-norm
            CUBLAS_CALL( hipblasSdot(cublasHandle, numCenters * numFeatures, deviceOldCenters, 1, deviceOldCenters, 1, &diff) );
            endFlag = diff < tolerance;
        }
    }

    // Copy the result back to host
    // Tranpose the deviceCenters back
    transposeMatrix(deviceCenters, numFeatures, numCenters);

    CUDA_CALL( hipMemcpy(centers, deviceCenters, sizeof(float) * numCenters * numFeatures, hipMemcpyDeviceToHost) );
    CUDA_CALL( hipMemcpy(membership, deviceMembership, sizeof(int) * numSamples, hipMemcpyDeviceToHost) );
    *numIterations = iterationCount;

    // Free all resources on GPU
    CUDA_CALL( hipFree(deviceX) );
    CUDA_CALL( hipFree(deviceCenters) );
    CUDA_CALL( hipFree(deviceMembership) );
    CUDA_CALL( hipFree(deviceOldCenters) );
    CUDA_CALL( hipFree(deviceSamplesCount) );

    // Free resources on CPU
    if (samplesCount) {
        free(samplesCount);
    }

    CUDA_CALL( hipblasDestroy(cublasHandle) );
    return;
}