#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <pudding/dimension_reduction.h>
#include <helper/helper_CUDA.h>
#include <helper/helper.cuh>

void _pcaGPU(const float* X, const int numSamples, const int numFeatures, const int numComponets, const float variancePercentage, float* principalComponets, float* principalAxes, float* variances) {
    /*
     * The GPU version of PCA
     * PCA can be implemented using the cuBLAS and cuSolver library.
     * 
     * PCA has four major steps:
     * 1. Compute the mean of X.
     * 2. Perform SVD on the centered data X - mean.
     * 3. Select the number of components using either numComponents of variancePercentage.
     * 4. Obtain the principal components, set the return results
     */

    // Perform simple pre-condition check
    if (numComponets == -1) {
        assert (variancePercentage > 0 && variancePercentage < 1);
    } else {
        assert (numComponets < min(numSamples, numFeatures));
    }

    // Malloc space on GPU
    float* deviceX;
    float* deviceAllOneVec;
    float* deviceMeanVec; // The mean vector of X.
    float* deviceCenteredX; // The centered X.
    float* deviceWorkBuffer; // The work buffer for performing SVD using cusolver.
    float* deviceS; // The sorted non-zero singular values.
    float* deviceU; // The left singular matrix U.
    float* deviceV; // The right singular matrix V^T.
    float* deviceVariances; // The actual variances along principal directions.
    // Temporary array for transpose
    float* tempDeviceX;
    
    CUDA_CALL( hipMalloc(&deviceX, sizeof(float) * numSamples * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceAllOneVec, sizeof(float) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceMeanVec, sizeof(float) * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceCenteredX, sizeof(float) * numSamples * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceS, sizeof(float) * min(numSamples, numFeatures)) );
    CUDA_CALL( hipMalloc(&deviceU, sizeof(float) * numSamples * numSamples) );
    CUDA_CALL( hipMalloc(&deviceV, sizeof(float) * numFeatures * numFeatures) );
    CUDA_CALL( hipMalloc(&tempDeviceX, sizeof(float) * numSamples * numFeatures) );

    CUDA_CALL( hipMemset(deviceAllOneVec, 1, sizeof(float) * numSamples) );

    // Determine the block width
    const int BLOCKWIDTH = 1024;

    // Prepare the handle for cublas
    hipblasHandle_t cublasHandle = NULL;
    CUBLAS_CALL( hipblasCreate(&cublasHandle) );

    // Prepare useful constant for cublas calls
    const float one = 1.0, zero = 0.0, negOne = -1.0;

    // Transpose deviceX to enable coalesced memory access in the kernel
    CUDA_CALL( hipMemcpy(tempDeviceX, X, sizeof(float) * numSamples * numFeatures, hipMemcpyHostToDevice) );
    CUBLAS_CALL( hipblasSgeam(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, numSamples, numFeatures, &one, tempDeviceX, numFeatures, &zero, tempDeviceX, numSamples, deviceX, numSamples) );

    // PCA begins here:
    // 1. Compute the mean of X. X is of shape (numSample, numFeature), the mean vector is of shape (numFeature,)
    // We use the cublas call to perform a matrix-vector matrix multiplcation in order to perform the row reduction.
    float oneOverNumSamples = 1.0 / numSamples;
    CUBLAS_CALL( hipblasSgemv(cublasHandle, HIPBLAS_OP_T, numSamples, numFeatures, &oneOverNumSamples, deviceX, numSamples, deviceAllOneVec, one, &zero, deviceMeanVec, one) );

    // 2. Perform SVD on the centered data.
    // 2.1. Center the data
    wrapperMatrixVectorSubtraction(deviceX, numFeatures, numSamples, deviceMeanVec, deviceCenteredX);
    // 2.2. Perform SVD on deviceCenteredX
    // TODO: Maybe the default configuration is better?
    // Configuration of gesvdj
    const float tol = 1e-7;
    const int maxSweeps = 15;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    const int econ = 0;

    // Numerical results of gesvdj
    float residual = 0;
    int executedSweeps = 0;

    // Create the cusolver handle and bind a stream
    hipsolverHandle_t cusolverHandle = NULL;
    hipStream_t stream = NULL;
    CUSOLVER_CALL( hipsolverDnCreate(&cusolverHandle) );
    CUDA_CALL( hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) );
    CUSOLVER_CALL( hipsolverSetStream(cusolverHandle, stream) );

    // Set the configuration of gesvdj
    hipsolverGesvdjInfo_t gesvdParams = NULL;
    CUSOLVER_CALL( hipsolverDnCreateGesvdjInfo(&gesvdParams) );
    CUSOLVER_CALL( hipsolverDnXgesvdjSetTolerance(gesvdParams, tol) );
    CUSOLVER_CALL( hipsolverDnXgesvdjSetMaxSweeps(gesvdParams, maxSweeps) );

    // Prepare the work buffer
    float workBufferSize = 0.0;
    CUSOLVER_CALL( hipsolverDnSgesvdj_bufferSize(cusolverHandle, jobz, econ, numSamples, numFeatures, deviceCenteredX, numSamples, deviceS, deviceU, numSamples, deviceV, numFeatures, &workBufferSize, gesvdParams) );
    CUDA_CALL( hipMalloc(&deviceWorkBuffer, sizeof(float) * workBufferSize) );

    // Perform the actual SVD computation
    int info = 0;
    CUSOLVER_CALL( hipsolverDnSgesvdj(cusolverHandle, jobz, econ, numSamples, numFeatures, deviceCenteredX, numSamples, deviceS, deviceU, numSamples, deviceV, numFeatures, deviceWorkBuffer, workBufferSize, &info, gesvdParams) );

    // TODO: Do we need to synchronize device here?

    // 3. Select the number of components
    if (numComponets == -1) {
        // 3.1. In this case, we need to select the number of components such that the ratio of the accumulated variance goes above the required variancePercentage
        assert (false); // TODO: For simplicity, add this later.
    }

    // 4. Obtain the principal components and set the result
    // 4.1. Compute the variances using the singular values
    // TODO: write our own kernel that performs element-wise multiplication

    // Free GPU spaces
    CUDA_CALL( hipFree(deviceX) );
    CUDA_CALL( hipFree(deviceAllOneVec) );
    CUDA_CALL( hipFree(deviceMeanVec) );
    CUDA_CALL( hipFree(deviceCenteredX) );
    CUDA_CALL( hipFree(deviceWorkBuffer) );
    CUDA_CALL( hipFree(deviceS) );
    CUDA_CALL( hipFree(deviceU) );
    CUDA_CALL( hipFree(deviceV) );
    CUDA_CALL( hipFree(tempDeviceX) );

    CUBLAS_CALL( hipblasDestroy(cublasHandle) );
    CUDA_CALL( hipStreamDestroy(stream) );
    CUSOLVER_CALL( hipsolverDnDestroy(cusolverHandle) );
    CUSOLVER_CALL( hipsolverDnDestroyGesvdjInfo(gesvdParams) );
}