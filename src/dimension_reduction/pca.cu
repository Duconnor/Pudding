#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include <pudding/dimension_reduction.h>
#include <helper/helper_CUDA.h>
#include <helper/helper.cuh>

void _pcaGPU(const float* X, const int numSamples, const int numFeatures, const int numComponents, const float variancePercentage, float* principalComponets, float* principalAxes, float* variances, int* numComponentsChosen) {
    /*
     * The GPU version of PCA
     * PCA can be implemented using the cuBLAS and cuSolver library.
     * 
     * PCA has four major steps:
     * 1. Compute the mean of X.
     * 2. Perform SVD on the centered data X - mean.
     * 3. Select the number of components using either numComponents of variancePercentage.
     * 4. Obtain the principal components, set the return results
     */

    // Perform simple pre-condition check
    if (numComponents == -1) {
        assert (variancePercentage > 0 && variancePercentage < 1);
    } else {
        assert (numComponents < min(numSamples, numFeatures));
    }

    // Malloc space on GPU
    float* deviceX;
    float* deviceAllOneVec;
    float* deviceMeanVec; // The mean vector of X.
    float* deviceCenteredX; // The centered X.
    float* deviceWorkBuffer; // The work buffer for performing SVD using cusolver.
    float* deviceS; // The sorted non-zero singular values.
    float* deviceU; // The left singular matrix U.
    float* deviceV; // The right singular matrix V^T.
    float* deviceVariances; // The actual variances along principal directions.
    float* devicePrincipalComponets; // The principal components (i.e. the lower dimensional representation of the original data)
    
    CUDA_CALL( hipMalloc(&deviceX, sizeof(float) * numSamples * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceAllOneVec, sizeof(float) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceMeanVec, sizeof(float) * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceCenteredX, sizeof(float) * numSamples * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceS, sizeof(float) * min(numSamples, numFeatures)) );
    CUDA_CALL( hipMalloc(&deviceU, sizeof(float) * numSamples * numSamples) );
    CUDA_CALL( hipMalloc(&deviceV, sizeof(float) * numFeatures * numFeatures) );
    CUDA_CALL( hipMalloc(&deviceVariances, sizeof(float) * min(numFeatures, numSamples)) );

    CUDA_CALL( hipMemset(deviceAllOneVec, 1, sizeof(float) * numSamples) );

    // Prepare the handle for cublas
    hipblasHandle_t cublasHandle = NULL;
    CUBLAS_CALL( hipblasCreate(&cublasHandle) );

    // Prepare useful constant for cublas calls
    const float one = 1.0, zero = 0.0;

    // Transpose deviceX to enable coalesced memory access in the kernel
    transposeMatrix(deviceX, numSamples, numFeatures);

    // PCA begins here:
    // 1. Compute the mean of X. X is of shape (numSample, numFeature), the mean vector is of shape (numFeature,)
    // We use the cublas call to perform a matrix-vector matrix multiplcation in order to perform the row reduction.
    float oneOverNumSamples = 1.0 / numSamples;
    CUBLAS_CALL( hipblasSgemv(cublasHandle, HIPBLAS_OP_T, numSamples, numFeatures, &oneOverNumSamples, deviceX, numSamples, deviceAllOneVec, one, &zero, deviceMeanVec, one) );

    // 2. Perform SVD on the centered data.
    // 2.1. Center the data
    wrapperMatrixVectorSubtraction(deviceX, numFeatures, numSamples, deviceMeanVec, deviceCenteredX);
    // 2.2. Perform SVD on deviceCenteredX
    // TODO: Maybe the default configuration is better?
    // Configuration of gesvdj
    const float tol = 1e-7;
    const int maxSweeps = 15;
    const hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR;
    const int econ = 0;

    // Create the cusolver handle and bind a stream
    hipsolverHandle_t cusolverHandle = NULL;
    hipStream_t stream = NULL;
    CUSOLVER_CALL( hipsolverDnCreate(&cusolverHandle) );
    CUDA_CALL( hipStreamCreateWithFlags(&stream, hipStreamNonBlocking) );
    CUSOLVER_CALL( hipsolverSetStream(cusolverHandle, stream) );

    // Set the configuration of gesvdj
    hipsolverGesvdjInfo_t gesvdParams = NULL;
    CUSOLVER_CALL( hipsolverDnCreateGesvdjInfo(&gesvdParams) );
    CUSOLVER_CALL( hipsolverDnXgesvdjSetTolerance(gesvdParams, tol) );
    CUSOLVER_CALL( hipsolverDnXgesvdjSetMaxSweeps(gesvdParams, maxSweeps) );

    // Prepare the work buffer
    int workBufferSize = 0;
    CUSOLVER_CALL( hipsolverDnSgesvdj_bufferSize(cusolverHandle, jobz, econ, numSamples, numFeatures, deviceCenteredX, numSamples, deviceS, deviceU, numSamples, deviceV, numFeatures, &workBufferSize, gesvdParams) );
    CUDA_CALL( hipMalloc(&deviceWorkBuffer, sizeof(float) * workBufferSize) );

    // Perform the actual SVD computation
    int info = 0;
    CUSOLVER_CALL( hipsolverDnSgesvdj(cusolverHandle, jobz, econ, numSamples, numFeatures, deviceCenteredX, numSamples, deviceS, deviceU, numSamples, deviceV, numFeatures, deviceWorkBuffer, workBufferSize, &info, gesvdParams) );

    // TODO: Do we need to synchronize device here?

    // 3. Select the number of components
    if (numComponents == -1) {
        // 3.1. In this case, we need to select the number of components such that the ratio of the accumulated variance goes above the required variancePercentage
        assert (false); // TODO: For simplicity, add this later.
    } else {
        *numComponentsChosen = numComponents;
    }

    // 4. Obtain the principal components and set the result
    // 4.1. Compute the variances using the singular values
    wrapperVectorVectorElementWiseMultiplication(deviceS, deviceS, min(numFeatures, numSamples), 1.0 / (numSamples - 1), deviceVariances);
    // 4.2. Copy the first numComponents elements from deviceVariances to variances
    CUDA_CALL( hipMemcpy(variances, deviceVariances, sizeof(float) * (*numComponentsChosen), hipMemcpyDeviceToHost) );
    // 4.3. Copy the first numComponents columns from V to principalAxes
    // Since cublas uses column-major storage, the column of V is stored continuously
    // Therefore, we can simply copy the first numComponents * numFeatures * sizeof(float) here :)
    CUDA_CALL( hipMemcpy(principalAxes, deviceV, sizeof(float) * (*numComponentsChosen) * numFeatures, hipMemcpyDeviceToHost) );
    // 4.4. Obtain the principal components by performing U * S
    CUDA_CALL( hipMalloc(&devicePrincipalComponets, sizeof(float) * numSamples * (*numComponentsChosen)) );
    CUDA_CALL( hipblasSdgmm(cublasHandle, HIPBLAS_SIDE_RIGHT, numSamples, *numComponentsChosen, deviceU, numSamples, deviceS, one, devicePrincipalComponets, numSamples) );
    transposeMatrix(devicePrincipalComponets, *numComponentsChosen, numSamples);
    CUDA_CALL( hipMemcpy(principalComponets, devicePrincipalComponets, sizeof(float) * numSamples * *numComponentsChosen, hipMemcpyDeviceToHost) );

    // Free GPU spaces
    CUDA_CALL( hipFree(deviceX) );
    CUDA_CALL( hipFree(deviceAllOneVec) );
    CUDA_CALL( hipFree(deviceMeanVec) );
    CUDA_CALL( hipFree(deviceCenteredX) );
    CUDA_CALL( hipFree(deviceWorkBuffer) );
    CUDA_CALL( hipFree(deviceS) );
    CUDA_CALL( hipFree(deviceU) );
    CUDA_CALL( hipFree(deviceV) );
    CUDA_CALL( hipFree(deviceVariances) );
    CUDA_CALL( hipFree(devicePrincipalComponets) );

    CUBLAS_CALL( hipblasDestroy(cublasHandle) );
    CUDA_CALL( hipStreamDestroy(stream) );
    CUSOLVER_CALL( hipsolverDnDestroy(cusolverHandle) );
    CUSOLVER_CALL( hipsolverDnDestroyGesvdjInfo(gesvdParams) );
}