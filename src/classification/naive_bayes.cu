#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <helper/helper_CUDA.h>
#include <helper/helper.cuh>

void _naiveBayesMultinomialFitGPU(const float* X, const int* y, const int numSamples, const int vocabularySize, const int numClasses, const float alpha, float* classProbability, float* wordProbability) {
    /*
     * This function fits a Naive Bayes model, it involves these steps:
     * 1. For every class:
     *  1.1. Find the total number of occurance for each word in the vocabulary plus the alpha (Laplace smoothing)
     *  1.2. Find the total number of samples belonging to this class.
     *  1.3. Divide the number of samples in this class by the total number of samples -> the class probability.
     *  1.4. Divide the total number of occurance of words by the number of samples in this class -> the word probability.
     * 
     */

    float one = 1.0, zero = 0.0;

    // Malloc space on GPU
    float* deviceX;
    int* deviceY;
    float* deviceWordProbability;
    float* deviceMaskVec; // This mask vec is used to count the occurance of each word, as well as the number of samples belonging to each class
    float* deviceAllOneVec; // This is a all one vector, used to pre-compute the number of words in each sample
    float* deviceWordsCount;

    CUDA_CALL( hipMalloc(&deviceX, sizeof(float) * numSamples * vocabularySize) );
    CUDA_CALL( hipMalloc(&deviceY, sizeof(int) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceWordProbability, sizeof(float) * numClasses * vocabularySize) );
    CUDA_CALL( hipMalloc(&deviceMaskVec, sizeof(float) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceAllOneVec, sizeof(float) * vocabularySize) );
    CUDA_CALL( hipMalloc(&deviceWordsCount, sizeof(float) * numSamples) );

    CUDA_CALL( hipMemcpy(deviceX, X, sizeof(float) * numSamples * vocabularySize, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceY, y, sizeof(int) * numSamples, hipMemcpyHostToDevice) );
    wrapperInitializeAllElementsToXKernel(deviceWordProbability, one, numClasses * vocabularySize); // It's important here to initialize all elements to one so we can do all things in one blas operation
    wrapperInitializeAllElementsToXKernel(deviceAllOneVec, one, vocabularySize);

    // Prepare cublas handle
    hipblasHandle_t cublasHandle;
    CUBLAS_CALL( hipblasCreate(&cublasHandle) );

    // Pre-compute the number of words in each sample
    // This can be done via a matrix vector multiplcation
    CUBLAS_CALL( hipblasSgemv(cublasHandle, HIPBLAS_OP_T, vocabularySize, numSamples, &one, deviceX, vocabularySize, deviceAllOneVec, one, &zero, deviceWordsCount, one) );

    // Start the main loop (loop for every class)
    for (int classIdx = 0; classIdx < numClasses; classIdx++) {
        // Generate the mask vector first
        wrapperGenerateMaskVectorKernel(deviceY, classIdx, numSamples, deviceMaskVec);
        // Multiply the mask vector to the pre-computed words count vector yields the total number of words in samples belonging to this class
        float numWordsThisClass = 0.0;
        CUBLAS_CALL( hipblasSdot(cublasHandle, numSamples, deviceWordsCount, one, deviceMaskVec, one, &numWordsThisClass) );
        float numSamplesThisClass = 0.0;
        CUBLAS_CALL( hipblasSasum(cublasHandle, numSamples, deviceMaskVec, one, &numSamplesThisClass) );
        // Multiply deviceX with deivceMaskVec to obtain the total number of occurance for each word in samples belonging to this class
        // Further divide the result by numWordsThisClass results in the word probability estimation
        float denominator = 1.0 / (numWordsThisClass + (vocabularySize * alpha));
        float beta = alpha * denominator;
        CUBLAS_CALL( hipblasSgemv(cublasHandle, HIPBLAS_OP_N, vocabularySize, numSamples, &denominator, deviceX, vocabularySize, deviceMaskVec, one, &beta, deviceWordProbability + (classIdx * vocabularySize), one) );
        classProbability[classIdx] = numSamplesThisClass / numSamples;
    }

    // Copy the result back to host
    CUDA_CALL( hipMemcpy(wordProbability, deviceWordProbability, sizeof(float) * numClasses * vocabularySize, hipMemcpyDeviceToHost) );

    // Free resources
    CUDA_CALL( hipFree(deviceX) );
    CUDA_CALL( hipFree(deviceY) );
    CUDA_CALL( hipFree(deviceWordProbability) );
    CUDA_CALL( hipFree(deviceMaskVec) );

    CUBLAS_CALL( hipblasDestroy(cublasHandle) );
}
