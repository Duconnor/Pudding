#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <helper/helper_CUDA.h>
#include <helper/helper.cuh>

void _naiveBayesMultinomialFitGPU(const float* X, const int* y, const int numSamples, const int vocabularySize, const int numClasses, const float alpha, float* classProbability, float* wordProbability) {
    /*
     * This function fits a Naive Bayes model, it involves these steps:
     * 1. For every class:
     *  1.1. Find the total number of occurance for each word in the vocabulary plus the alpha (Laplace smoothing)
     *  1.2. Find the total number of samples belonging to this class.
     *  1.3. Divide the number of samples in this class by the total number of samples -> the class probability.
     *  1.4. Divide the total number of occurance of words by the number of samples in this class -> the word probability.
     * 
     */

    float one = 1.0, zero = 0.0;

    // Malloc space on GPU
    float* deviceX;
    int* deviceY;
    float* deviceWordProbability;
    float* deviceMaskVec; // This mask vec is used to count the occurance of each word, as well as the number of samples belonging to each class
    float* deviceAllOneVec; // This is a all one vector, used to pre-compute the number of words in each sample
    float* deviceWordsCount;

    CUDA_CALL( hipMalloc(&deviceX, sizeof(float) * numSamples * vocabularySize) );
    CUDA_CALL( hipMalloc(&deviceY, sizeof(int) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceWordProbability, sizeof(float) * numClasses * vocabularySize) );
    CUDA_CALL( hipMalloc(&deviceMaskVec, sizeof(float) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceAllOneVec, sizeof(float) * vocabularySize) );
    CUDA_CALL( hipMalloc(&deviceWordsCount, sizeof(float) * numSamples) );

    CUDA_CALL( hipMemcpy(deviceX, X, sizeof(float) * numSamples * vocabularySize, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceY, y, sizeof(int) * numSamples, hipMemcpyHostToDevice) );
    wrapperInitializeAllElementsToXKernel(deviceWordProbability, one, numClasses * vocabularySize); // It's important here to initialize all elements to one so we can do all things in one blas operation
    wrapperInitializeAllElementsToXKernel(deviceAllOneVec, one, vocabularySize);

    // Prepare cublas handle
    hipblasHandle_t cublasHandle;
    CUBLAS_CALL( hipblasCreate(&cublasHandle) );

    // Pre-compute the number of words in each sample
    // This can be done via a matrix vector multiplcation
    CUBLAS_CALL( hipblasSgemv(cublasHandle, HIPBLAS_OP_T, vocabularySize, numSamples, &one, deviceX, vocabularySize, deviceAllOneVec, one, &zero, deviceWordsCount, one) );

    // Start the main loop (loop for every class)
    for (int classIdx = 0; classIdx < numClasses; classIdx++) {
        // Generate the mask vector first
        wrapperGenerateMaskVectorKernel(deviceY, classIdx, numSamples, deviceMaskVec);
        // Multiply the mask vector to the pre-computed words count vector yields the total number of words in samples belonging to this class
        float numWordsThisClass = 0.0;
        CUBLAS_CALL( hipblasSdot(cublasHandle, numSamples, deviceWordsCount, one, deviceMaskVec, one, &numWordsThisClass) );
        float numSamplesThisClass = 0.0;
        CUBLAS_CALL( hipblasSasum(cublasHandle, numSamples, deviceMaskVec, one, &numSamplesThisClass) );
        // Multiply deviceX with deivceMaskVec to obtain the total number of occurance for each word in samples belonging to this class
        // Further divide the result by numWordsThisClass results in the word probability estimation
        float denominator = 1.0 / (numWordsThisClass + (vocabularySize * alpha));
        float beta = alpha * denominator;
        CUBLAS_CALL( hipblasSgemv(cublasHandle, HIPBLAS_OP_N, vocabularySize, numSamples, &denominator, deviceX, vocabularySize, deviceMaskVec, one, &beta, deviceWordProbability + (classIdx * vocabularySize), one) );
        classProbability[classIdx] = numSamplesThisClass / numSamples;
    }

    // Copy the result back to host
    CUDA_CALL( hipMemcpy(wordProbability, deviceWordProbability, sizeof(float) * numClasses * vocabularySize, hipMemcpyDeviceToHost) );

    // Free resources
    CUDA_CALL( hipFree(deviceX) );
    CUDA_CALL( hipFree(deviceY) );
    CUDA_CALL( hipFree(deviceWordProbability) );
    CUDA_CALL( hipFree(deviceMaskVec) );

    CUBLAS_CALL( hipblasDestroy(cublasHandle) );
}

void _naiveBayesMultinomialPredictGPU(const float* X, const float* classProbability, const float* wordProbability, const int numSamples, const int vocabularySize, const int numClasses, int* predictions) {
    /*
     * This function make predictions using the fitted Naive Bayes model.
     * To make predictions, we simply need to calculate the posterior probability p(y|x), which can be expanded as:
     * p(y|x) = p(x|y) * p(y) / p(x) \propto p(x|y) * p(y)
     * p(y) is exactly what we have in classProbability, therefore the key is to compute p(x|y). Since we use multinomial event model, p(x|y) can be written as:
     * p(x|y) = \prod_{i=0}^{|V| - 1} p(i|y)^{x_i}, where p(i|y) is what we have in wordProbability.
     * It's not hard to see there invovles multiplication, which is difficult to implement in CUDA. However, we can apply the simple log trick and instead compute the log (p(x|y) * p(y)) = log p(x|y) + log p(y), where we further have:
     * log p(x|y) = \sum_{i=1}^{|V|-1} (x_i * log p(i|y)). This can be implemented using a blas operation.
     * 
     * Specifically, this function takes the following steps:
     * 1. Apply the logarithmic function to elements in classProbability and wordProbability. -> Our own kernels.
     * 2. Perform the matrix multiplication between X and log(wordProbability). -> We can use cublas.
     * 3. Add the result matrix from above to the classProbability vector (a broadcast). -> Our own kernels.
     * 4. The result matrix is the posterior probability matrix, we then find the argmax for each sample, which is the predictions we made. -> Our own kernels.
     * 
     */

    float one = 1.0, zero = 0.0;

    // Malloc space on GPU
    float* deviceX;
    float* deviceClassProbability;
    float* deviceWordProbability;
    float* devicePostProbability;
    int* devicePredictions;
    float* deviceMaxPostProbability;

    CUDA_CALL( hipMalloc(&deviceX, sizeof(float) * numSamples * vocabularySize) );
    CUDA_CALL( hipMalloc(&deviceClassProbability, sizeof(float) * numClasses) );
    CUDA_CALL( hipMalloc(&deviceWordProbability, sizeof(float) * numClasses * vocabularySize) );
    CUDA_CALL( hipMalloc(&devicePostProbability, sizeof(float) * numSamples * numClasses) );
    CUDA_CALL( hipMalloc(&devicePredictions, sizeof(int) * numSamples) );
    CUDA_CALL( hipMalloc(&deviceMaxPostProbability, sizeof(float) * numSamples) );

    CUDA_CALL( hipMemcpy(deviceX, X, sizeof(float) * numSamples * vocabularySize, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceClassProbability, classProbability, sizeof(float) * numClasses, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceWordProbability, wordProbability, sizeof(float) * numClasses * vocabularySize, hipMemcpyHostToDevice) );

    // Prepare for the cublas handle
    hipblasHandle_t cublasHandle;
    CUBLAS_CALL( hipblasCreate(&cublasHandle) );

    // 1. Apply the log function to each elements in the class probabilit and word probabilit matrix
    wrapperApplyUnaryFunctionKernel(deviceClassProbability, numClasses, LOG);
    wrapperApplyUnaryFunctionKernel(deviceWordProbability, numClasses * vocabularySize, LOG);

    // 2. The matrix multiplication between X and log(wordProbability)
    CUBLAS_CALL( hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, numSamples, numClasses, vocabularySize, &one, deviceX, vocabularySize, deviceWordProbability, vocabularySize, &zero, devicePostProbability, numSamples) );

    // 3. Add the class probability matrix
    wrapperMatrixVectorAddition(devicePostProbability, numClasses, numSamples, deviceClassProbability, one, devicePostProbability);
    transposeMatrix(devicePostProbability, numClasses, numSamples);

    // 4. Select the argmax
    wrapperMatrixArgMaxRowKernel(devicePostProbability, numSamples, numClasses, deviceMaxPostProbability, devicePredictions);

    // Copy the result back to host
    CUDA_CALL( hipMemcpy(predictions, devicePredictions, sizeof(int) * numSamples, hipMemcpyDeviceToHost) );

    // Free all resources
    CUDA_CALL( hipFree(deviceX) );
    CUDA_CALL( hipFree(deviceClassProbability) );
    CUDA_CALL( hipFree(deviceWordProbability) );
    CUDA_CALL( hipFree(devicePostProbability) );
    CUDA_CALL( hipFree(deviceMaxPostProbability) );
    CUDA_CALL( hipFree(devicePredictions) );

    CUBLAS_CALL( hipblasDestroy(cublasHandle) );
}
