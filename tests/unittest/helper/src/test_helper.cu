#define CATCH_CONFIG_MAIN
#include <cstdlib>
#include <vector>

#include <catch2/catch.hpp>
#include <hip/hip_runtime.h>

#include <helper/helper.cuh>
#include <helper/helper_CUDA.h>

#include <helper.h>

TEST_CASE ("Test matrix vector addition kernel", "[matrix-vector-addition]") {
    // Prepare the test data
    const int numRow = 3;
    const int numCol = 4;
    std::vector<float> matrix = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0};
    std::vector<float> vector = {5.0, 5.0, 5.0};
    float scale = -1.0;

    std::vector<float> expectedRes = {-4.0, -3.0, -2.0, -1.0, 0.0, 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0};

    // Copy data to device
    float* deviceMatrix = NULL;
    float* deviceVector = NULL;
    float* deviceRes = NULL;

    CUDA_CALL( hipMalloc(&deviceMatrix, sizeof(float) * numRow * numCol) );
    CUDA_CALL( hipMalloc(&deviceVector, sizeof(float) * numRow) );
    CUDA_CALL( hipMalloc(&deviceRes, sizeof(float) * numRow * numCol) );

    CUDA_CALL( hipMemcpy(deviceMatrix, matrix.data(), sizeof(float) * numRow * numCol, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceVector, vector.data(), sizeof(float) * numRow, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperMatrixVectorAddition(deviceMatrix, numRow, numCol, deviceVector, scale, deviceRes);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numRow * numCol);
    CUDA_CALL( hipMemcpy(res, deviceRes, sizeof(float) * numRow * numCol, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + (numRow * numCol));
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceMatrix) );
    CUDA_CALL( hipFree(deviceVector) );
    CUDA_CALL( hipFree(deviceRes) );
}

TEST_CASE ("Test vector vector element wise multiplication", "[vector-vector-element-wise-multiplication]") {
    // Prepare the test data
    const int numElements = 6;
    std::vector<float> vecOne = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0};
    std::vector<float> vecTwo = {7.0, 8.0, 9.0, 10.0, 11.0, 12.0};
    float scale = 2.0;

    std::vector<float> expectedRes = {14.0, 32.0, 54.0, 80.0, 110.0, 144.0};

    // Copy data to device
    float* deviceVecOne = NULL;
    float* deviceVecTwo = NULL;
    float* deviceRes = NULL;

    CUDA_CALL( hipMalloc(&deviceVecOne, sizeof(float) * numElements) );
    CUDA_CALL( hipMalloc(&deviceVecTwo, sizeof(float) * numElements) );
    CUDA_CALL( hipMalloc(&deviceRes, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceVecOne, vecOne.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceVecTwo, vecTwo.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperVectorVectorElementWiseMultiplication(deviceVecOne, deviceVecTwo, numElements, scale, deviceRes);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(res, deviceRes, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + numElements);
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceVecOne) );
    CUDA_CALL( hipFree(deviceVecTwo) );
    CUDA_CALL( hipFree(deviceRes) );
}

TEST_CASE ("Test matrix transpose", "[matrix-transpose]") {
    // Prepare the test data
    const int numElements = 12;
    std::vector<float> matrix = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0};

    std::vector<float> expectedRes = {1.0, 4.0, 7.0, 10.0, 2.0, 5.0, 8.0, 11.0, 3.0, 6.0, 9.0, 12.0};

    // Copy data to device
    float* deviceMatrix = NULL;

    CUDA_CALL( hipMalloc(&deviceMatrix, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceMatrix, matrix.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    transposeMatrix(deviceMatrix, 4, 3);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(res, deviceMatrix, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + numElements);
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceMatrix) );
}

TEST_CASE ("Test array initialization", "[array-initialization]") {
    // Prepare the test data
    const int numElements = 12;
    const float targetValue = 233;
    std::vector<float> vec(numElements, 0);

    std::vector<float> expectedRes(numElements, targetValue);

    // Copy data to device
    float* deviceVec = NULL;

    CUDA_CALL( hipMalloc(&deviceVec, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceVec, vec.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperInitializeAllElementsToXKernel(deviceVec, targetValue, numElements);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(res, deviceVec, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + numElements);
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceVec) );
}

TEST_CASE ("Test pair wise euclidean distance computation", "[pair-wise-euclidean-distance]") {
    // Prepare the test data
    const int numExamplesRef = 3;
    const int numExamplesQuery = 2;
    const int numFeatures = 2;

    std::vector<std::vector<float>> refX = {{0, 2, -1}, {1, 1, -2}};
    std::vector<std::vector<float>> queryX = {{0, 2}, {1, -1}};

    std::vector<std::vector<float>> expectedDist = {{0, 8}, {4, 4}, {10, 10}};

    // Copy data to device
    float* deviceRefX;
    float* deviceQueryX;

    CUDA_CALL( hipMalloc(&deviceRefX, sizeof(float) * numFeatures * numExamplesRef) );
    CUDA_CALL( hipMalloc(&deviceQueryX, sizeof(float) * numFeatures * numExamplesQuery) );

    CUDA_CALL( hipMemcpy(deviceRefX, flatten(refX).data(), sizeof(float) * numFeatures * numExamplesRef, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceQueryX, flatten(queryX).data(), sizeof(float) * numFeatures * numExamplesQuery, hipMemcpyHostToDevice) );

    // Prepare for output
    float* deviceDist;

    CUDA_CALL( hipMalloc(&deviceDist, sizeof(float) * numExamplesRef * numExamplesQuery) );

    // Call the function
    wrapperComputePairwiseEuclideanDistanceKerenl(deviceRefX, deviceQueryX, numExamplesRef, numExamplesQuery, numFeatures, deviceDist);

    // Copy the output back to host
    float* dist = (float*)malloc(sizeof(float) * numExamplesRef * numExamplesQuery);

    CUDA_CALL( hipMemcpy(dist, deviceDist, sizeof(float) * numExamplesRef * numExamplesQuery, hipMemcpyDeviceToHost) );

    std::vector<float> vecDist(dist, dist + (numExamplesRef * numExamplesQuery));

    // Check
    REQUIRE_THAT(vecDist, Catch::Approx(flatten(expectedDist)));

    // Free resources
    CUDA_CALL( hipFree(deviceRefX) );
    CUDA_CALL( hipFree(deviceQueryX) );
    if (dist) {
        free(dist);
    }
}

TEST_CASE ("Test mask vector generation", "[mask-generation]") {
    // Prepare the test data
    const int numElements = 8;
    const int targetLabel = 2;
    std::vector<int> labelVec = {0, 1, 2, 1, 2, 0, 2, 10};

    std::vector<float> expectedMaskVec = {0, 0, 1, 0, 1, 0, 1, 0};

    // Copy data to device and also prepare space for result
    int* deviceLabelVec = NULL;
    float* deviceMaskVec = NULL;

    CUDA_CALL( hipMalloc(&deviceLabelVec, sizeof(int) * numElements) );
    CUDA_CALL( hipMalloc(&deviceMaskVec, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceLabelVec, labelVec.data(), sizeof(int) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperGenerateMaskVectorKernel(deviceLabelVec, targetLabel, numElements, deviceMaskVec);

    // Copy data back to host
    float* maskVec = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(maskVec, deviceMaskVec, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecMaskVec(maskVec, maskVec + numElements);
    REQUIRE_THAT(vecMaskVec, Catch::Approx(expectedMaskVec));

    if (maskVec) {
        free(maskVec);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceLabelVec) );
    CUDA_CALL( hipFree(deviceMaskVec) );
}
