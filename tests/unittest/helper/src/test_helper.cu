#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include <cstdlib>
#include <vector>

#include <catch2/catch.hpp>
#include <hip/hip_runtime.h>

#include <helper/helper.cuh>
#include <helper/helper_CUDA.h>

#include <helper.h>

TEST_CASE ("Test matrix vector addition kernel", "[matrix-vector-addition]") {
    // Prepare the test data
    const int numRow = 3;
    const int numCol = 4;
    std::vector<float> matrix = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0};
    std::vector<float> vector = {5.0, 5.0, 5.0};
    float scale = -1.0;

    std::vector<float> expectedRes = {-4.0, -3.0, -2.0, -1.0, 0.0, 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0};

    // Copy data to device
    float* deviceMatrix = NULL;
    float* deviceVector = NULL;
    float* deviceRes = NULL;

    CUDA_CALL( hipMalloc(&deviceMatrix, sizeof(float) * numRow * numCol) );
    CUDA_CALL( hipMalloc(&deviceVector, sizeof(float) * numRow) );
    CUDA_CALL( hipMalloc(&deviceRes, sizeof(float) * numRow * numCol) );

    CUDA_CALL( hipMemcpy(deviceMatrix, matrix.data(), sizeof(float) * numRow * numCol, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceVector, vector.data(), sizeof(float) * numRow, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperMatrixVectorAddition(deviceMatrix, numRow, numCol, deviceVector, scale, deviceRes);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numRow * numCol);
    CUDA_CALL( hipMemcpy(res, deviceRes, sizeof(float) * numRow * numCol, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + (numRow * numCol));
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceMatrix) );
    CUDA_CALL( hipFree(deviceVector) );
    CUDA_CALL( hipFree(deviceRes) );
}

TEST_CASE ("Test vector vector element wise multiplication", "[vector-vector-element-wise-multiplication]") {
    // Prepare the test data
    const int numElements = 6;
    std::vector<float> vecOne = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0};
    std::vector<float> vecTwo = {7.0, 8.0, 9.0, 10.0, 11.0, 12.0};
    float scale = 2.0;

    std::vector<float> expectedRes = {14.0, 32.0, 54.0, 80.0, 110.0, 144.0};

    // Copy data to device
    float* deviceVecOne = NULL;
    float* deviceVecTwo = NULL;
    float* deviceRes = NULL;

    CUDA_CALL( hipMalloc(&deviceVecOne, sizeof(float) * numElements) );
    CUDA_CALL( hipMalloc(&deviceVecTwo, sizeof(float) * numElements) );
    CUDA_CALL( hipMalloc(&deviceRes, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceVecOne, vecOne.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceVecTwo, vecTwo.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperVectorVectorElementWiseMultiplication(deviceVecOne, deviceVecTwo, numElements, scale, deviceRes);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(res, deviceRes, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + numElements);
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceVecOne) );
    CUDA_CALL( hipFree(deviceVecTwo) );
    CUDA_CALL( hipFree(deviceRes) );
}

TEST_CASE ("Test matrix transpose", "[matrix-transpose]") {
    // Prepare the test data
    const int numElements = 12;
    std::vector<float> matrix = {1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0, 11.0, 12.0};

    std::vector<float> expectedRes = {1.0, 4.0, 7.0, 10.0, 2.0, 5.0, 8.0, 11.0, 3.0, 6.0, 9.0, 12.0};

    // Copy data to device
    float* deviceMatrix = NULL;

    CUDA_CALL( hipMalloc(&deviceMatrix, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceMatrix, matrix.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    transposeMatrix(deviceMatrix, 4, 3);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(res, deviceMatrix, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + numElements);
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceMatrix) );
}

TEST_CASE ("Test array initialization", "[array-initialization]") {
    // Prepare the test data
    const int numElements = 12;
    const float targetValue = 233;
    std::vector<float> vec(numElements, 0);

    std::vector<float> expectedRes(numElements, targetValue);

    // Copy data to device
    float* deviceVec = NULL;

    CUDA_CALL( hipMalloc(&deviceVec, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceVec, vec.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperInitializeAllElementsToXKernel(deviceVec, targetValue, numElements);

    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(res, deviceVec, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecRes(res, res + numElements);
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

    if (res) {
        free(res);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceVec) );
}

TEST_CASE ("Test pair wise euclidean distance computation", "[pair-wise-euclidean-distance]") {
    // Prepare the test data
    const int numExamplesRef = 3;
    const int numExamplesQuery = 2;
    const int numFeatures = 2;

    std::vector<std::vector<float>> refX = {{0, 2, -1}, {1, 1, -2}};
    std::vector<std::vector<float>> queryX = {{0, 2}, {1, -1}};

    std::vector<std::vector<float>> expectedDist = {{0, 8}, {4, 4}, {10, 10}};

    // Copy data to device
    float* deviceRefX;
    float* deviceQueryX;

    CUDA_CALL( hipMalloc(&deviceRefX, sizeof(float) * numFeatures * numExamplesRef) );
    CUDA_CALL( hipMalloc(&deviceQueryX, sizeof(float) * numFeatures * numExamplesQuery) );

    CUDA_CALL( hipMemcpy(deviceRefX, flatten(refX).data(), sizeof(float) * numFeatures * numExamplesRef, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpy(deviceQueryX, flatten(queryX).data(), sizeof(float) * numFeatures * numExamplesQuery, hipMemcpyHostToDevice) );

    // Prepare for output
    float* deviceDist;

    CUDA_CALL( hipMalloc(&deviceDist, sizeof(float) * numExamplesRef * numExamplesQuery) );

    // Call the function
    wrapperComputePairwiseEuclideanDistanceKerenl(deviceRefX, deviceQueryX, numExamplesRef, numExamplesQuery, numFeatures, deviceDist);

    // Copy the output back to host
    float* dist = (float*)malloc(sizeof(float) * numExamplesRef * numExamplesQuery);

    CUDA_CALL( hipMemcpy(dist, deviceDist, sizeof(float) * numExamplesRef * numExamplesQuery, hipMemcpyDeviceToHost) );

    std::vector<float> vecDist(dist, dist + (numExamplesRef * numExamplesQuery));

    // Check
    REQUIRE_THAT(vecDist, Catch::Approx(flatten(expectedDist)));

    // Free resources
    CUDA_CALL( hipFree(deviceRefX) );
    CUDA_CALL( hipFree(deviceQueryX) );
    if (dist) {
        free(dist);
    }
}

TEST_CASE ("Test mask vector generation", "[mask-generation]") {
    // Prepare the test data
    const int numElements = 8;
    const int targetLabel = 2;
    std::vector<int> labelVec = {0, 1, 2, 1, 2, 0, 2, 10};

    std::vector<float> expectedMaskVec = {0, 0, 1, 0, 1, 0, 1, 0};

    // Copy data to device and also prepare space for result
    int* deviceLabelVec = NULL;
    float* deviceMaskVec = NULL;

    CUDA_CALL( hipMalloc(&deviceLabelVec, sizeof(int) * numElements) );
    CUDA_CALL( hipMalloc(&deviceMaskVec, sizeof(float) * numElements) );

    CUDA_CALL( hipMemcpy(deviceLabelVec, labelVec.data(), sizeof(int) * numElements, hipMemcpyHostToDevice) );

    // Launche the kernel
    wrapperGenerateMaskVectorKernel(deviceLabelVec, targetLabel, numElements, deviceMaskVec);

    // Copy data back to host
    float* maskVec = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(maskVec, deviceMaskVec, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    // Assertions
    std::vector<float> vecMaskVec(maskVec, maskVec + numElements);
    REQUIRE_THAT(vecMaskVec, Catch::Approx(expectedMaskVec));

    if (maskVec) {
        free(maskVec);
    }
    // Free all resources
    CUDA_CALL( hipFree(deviceLabelVec) );
    CUDA_CALL( hipFree(deviceMaskVec) );
}

TEST_CASE ("Test compute log of every elements in a vector", "[element-wise-log]") {
    // Prepare the test data
    const int numElements = 10;
    std::vector<float> vec = {1.0, 2.0, 3.0, 4.0, 5.0, 1000.0, 101.0, 10.23, 0.34, 0.78};

    std::vector<float> expectedRes = {0, 0.69314718, 1.09861229, 1.38629436, 1.60943791, 6.90775528, 4.61512052, 2.32532458, -1.07880966, -0.24846136};

    // Copy the data to device
    float* deviceVec;
    CUDA_CALL( hipMalloc(&deviceVec, sizeof(float) * numElements) );
    CUDA_CALL( hipMemcpy(deviceVec, vec.data(), sizeof(float) * numElements, hipMemcpyHostToDevice) );

    // Launch the function
    wrapperApplyUnaryFunctionKernel(deviceVec, numElements, LOG);
    
    // Copy data back to host
    float* res = (float*)malloc(sizeof(float) * numElements);
    CUDA_CALL( hipMemcpy(res, deviceVec, sizeof(float) * numElements, hipMemcpyDeviceToHost) );

    std::vector<float> vecRes(res, res + numElements);

    // Check
    REQUIRE_THAT(vecRes, Catch::Approx(expectedRes));

}

TEST_CASE ("Test selecting the maximum element and its index from each row of a matrix on a small dataset", "[matrix-argmax-row-small]") {
    // Prepare the test data
    std::srand(0);

    const int numRows = 200;
    const int numCols = 100;

    std::vector<std::vector<float>> matrix(numRows, std::vector<float>(numCols, 0.0));
    std::vector<int> expectedMaxIdx(numRows, -1);
    std::vector<float> expectedMaxVal(numRows, -1.0);
    for (int i = 0; i < numRows; i++) {
        float currentMax = -FLT_MAX, currentMaxIdx = -1;
        for (int j = 0 ; j < numCols; j++) {
            float randVal = rand() / (float)(RAND_MAX / 1000000000.0);
            matrix[i][j] = randVal;
            if (randVal > currentMax) {
                currentMax = randVal;
                currentMaxIdx = j;
            }
        }
        expectedMaxIdx[i] = currentMaxIdx;
        expectedMaxVal[i] = currentMax;
    }

    // Copy the data to device
    float* deviceMatrix;
    float* deviceMaxVal;
    int* deviceMaxIdx;

    CUDA_CALL( hipMalloc(&deviceMatrix, sizeof(float) * numRows * numCols) );
    CUDA_CALL( hipMalloc(&deviceMaxVal, sizeof(float) * numRows) );
    CUDA_CALL( hipMalloc(&deviceMaxIdx, sizeof(int) * numRows) );

    CUDA_CALL( hipMemcpy(deviceMatrix, flatten(matrix).data(), sizeof(float) * numRows * numCols, hipMemcpyHostToDevice) );

    // Launch the function
    wrapperMatrixArgMaxRowKernel(deviceMatrix, numRows, numCols, deviceMaxVal, deviceMaxIdx);

    // Copy the result back to host and compare
    float* maxVal = (float*)malloc(sizeof(float) * numRows);
    int* maxIdx = (int*)malloc(sizeof(int) * numRows);

    CUDA_CALL( hipMemcpy(maxVal, deviceMaxVal, sizeof(float) * numRows, hipMemcpyDeviceToHost) );
    CUDA_CALL( hipMemcpy(maxIdx, deviceMaxIdx, sizeof(int) * numRows, hipMemcpyDeviceToHost) );

    std::vector<float> vecMaxVal(maxVal, maxVal + numRows);
    std::vector<int> vecMaxIdx(maxIdx, maxIdx + numRows);

    REQUIRE_THAT(vecMaxVal, Catch::Approx(expectedMaxVal));
    REQUIRE(vecMaxIdx == expectedMaxIdx);

    // Free resources
    CUDA_CALL( hipFree(deviceMatrix) );
    CUDA_CALL( hipFree(deviceMaxVal) );
    CUDA_CALL( hipFree(deviceMaxIdx) );

    if (maxVal) {
        free(maxVal);
    }
    if (maxIdx) {
        free(maxIdx);
    }
}


TEST_CASE ("Test selecting the maximum element and its index from each row of a matrix on a large dataset", "[matrix-argmax-row-large]") {
    // Prepare the test data
    std::srand(0);

    const int numRows = 70000;
    const int numCols = 2000;

    std::vector<std::vector<float>> matrix(numRows, std::vector<float>(numCols, 0.0));
    std::vector<int> expectedMaxIdx(numRows, -1);
    std::vector<float> expectedMaxVal(numRows, -1.0);
    for (int i = 0; i < numRows; i++) {
        float currentMax = -FLT_MAX, currentMaxIdx = -1;
        for (int j = 0 ; j < numCols; j++) {
            float randVal = rand() / (float)(RAND_MAX / 1000000000.0);
            matrix[i][j] = randVal;
            if (randVal > currentMax) {
                currentMax = randVal;
                currentMaxIdx = j;
            }
        }
        expectedMaxIdx[i] = currentMaxIdx;
        expectedMaxVal[i] = currentMax;
    }

    // Copy the data to device
    float* deviceMatrix;
    float* deviceMaxVal;
    int* deviceMaxIdx;

    CUDA_CALL( hipMalloc(&deviceMatrix, sizeof(float) * numRows * numCols) );
    CUDA_CALL( hipMalloc(&deviceMaxVal, sizeof(float) * numRows) );
    CUDA_CALL( hipMalloc(&deviceMaxIdx, sizeof(int) * numRows) );

    CUDA_CALL( hipMemcpy(deviceMatrix, flatten(matrix).data(), sizeof(float) * numRows * numCols, hipMemcpyHostToDevice) );

    // Launch the function
    wrapperMatrixArgMaxRowKernel(deviceMatrix, numRows, numCols, deviceMaxVal, deviceMaxIdx);

    // Copy the result back to host and compare
    float* maxVal = (float*)malloc(sizeof(float) * numRows);
    int* maxIdx = (int*)malloc(sizeof(int) * numRows);

    CUDA_CALL( hipMemcpy(maxVal, deviceMaxVal, sizeof(float) * numRows, hipMemcpyDeviceToHost) );
    CUDA_CALL( hipMemcpy(maxIdx, deviceMaxIdx, sizeof(int) * numRows, hipMemcpyDeviceToHost) );

    std::vector<float> vecMaxVal(maxVal, maxVal + numRows);
    std::vector<int> vecMaxIdx(maxIdx, maxIdx + numRows);

    // Given such many elements in the matrix (70000 * 2000 = 140000000) there will be several elements that are all maximum one. Therefore, it does not make any sense to compare the maximum index here.
    REQUIRE_THAT(vecMaxVal, Catch::Approx(expectedMaxVal));

    // Free resources
    CUDA_CALL( hipFree(deviceMatrix) );
    CUDA_CALL( hipFree(deviceMaxVal) );
    CUDA_CALL( hipFree(deviceMaxIdx) );

    if (maxVal) {
        free(maxVal);
    }
    if (maxIdx) {
        free(maxIdx);
    }
}

TEST_CASE ("Test selecting the maximum element and its index from each row of a matrix on negative elements", "[matrix-argmax-row-negative-elements]") {
    // Prepare the test data
    std::srand(0);

    const int numRows = 200;
    const int numCols = 100;

    std::vector<std::vector<float>> matrix(numRows, std::vector<float>(numCols, 0.0));
    std::vector<int> expectedMaxIdx(numRows, -1);
    std::vector<float> expectedMaxVal(numRows, -1.0);
    for (int i = 0; i < numRows; i++) {
        float currentMax = -FLT_MAX, currentMaxIdx = -1;
        for (int j = 0 ; j < numCols; j++) {
            float randVal = -(rand() / (float)(RAND_MAX / 1000000000.0));
            matrix[i][j] = randVal;
            if (randVal > currentMax) {
                currentMax = randVal;
                currentMaxIdx = j;
            }
        }
        expectedMaxIdx[i] = currentMaxIdx;
        expectedMaxVal[i] = currentMax;
    }

    // Copy the data to device
    float* deviceMatrix;
    float* deviceMaxVal;
    int* deviceMaxIdx;

    CUDA_CALL( hipMalloc(&deviceMatrix, sizeof(float) * numRows * numCols) );
    CUDA_CALL( hipMalloc(&deviceMaxVal, sizeof(float) * numRows) );
    CUDA_CALL( hipMalloc(&deviceMaxIdx, sizeof(int) * numRows) );

    CUDA_CALL( hipMemcpy(deviceMatrix, flatten(matrix).data(), sizeof(float) * numRows * numCols, hipMemcpyHostToDevice) );

    // Launch the function
    wrapperMatrixArgMaxRowKernel(deviceMatrix, numRows, numCols, deviceMaxVal, deviceMaxIdx);

    // Copy the result back to host and compare
    float* maxVal = (float*)malloc(sizeof(float) * numRows);
    int* maxIdx = (int*)malloc(sizeof(int) * numRows);

    CUDA_CALL( hipMemcpy(maxVal, deviceMaxVal, sizeof(float) * numRows, hipMemcpyDeviceToHost) );
    CUDA_CALL( hipMemcpy(maxIdx, deviceMaxIdx, sizeof(int) * numRows, hipMemcpyDeviceToHost) );

    std::vector<float> vecMaxVal(maxVal, maxVal + numRows);
    std::vector<int> vecMaxIdx(maxIdx, maxIdx + numRows);

    REQUIRE_THAT(vecMaxVal, Catch::Approx(expectedMaxVal));
    REQUIRE(vecMaxIdx == expectedMaxIdx);

    // Free resources
    CUDA_CALL( hipFree(deviceMatrix) );
    CUDA_CALL( hipFree(deviceMaxVal) );
    CUDA_CALL( hipFree(deviceMaxIdx) );

    if (maxVal) {
        free(maxVal);
    }
    if (maxIdx) {
        free(maxIdx);
    }
}
